#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include "implicit/gpu/convert.cuh"
#include "implicit/gpu/dot.cuh"
#include "implicit/gpu/matrix.h"
#include "implicit/gpu/utils.h"

namespace implicit {
namespace gpu {
template <typename T>
Vector<T>::Vector(int size, const T *host_data)
    : size(size),
      storage(new rmm::device_uvector<T>(size, rmm::cuda_stream_view())),
      data(storage->data()) {
  if (host_data) {
    CHECK_CUDA(
        hipMemcpy(data, host_data, size * sizeof(T), hipMemcpyHostToDevice));
  }
}

template <typename T> void Vector<T>::to_host(T *out) const {
  CHECK_CUDA(hipMemcpy(out, data, size * sizeof(T), hipMemcpyDeviceToHost));
}

template struct Vector<char>;
template struct Vector<int>;
template struct Vector<float>;

Matrix::Matrix(const Matrix &other, int rowid)
    : rows(1), cols(other.cols), data(other.at(rowid * other.cols)),
      storage(other.storage), itemsize(other.itemsize) {
  if (rowid >= other.rows) {
    throw std::invalid_argument("row index out of bounds for matrix");
  }
}

Matrix::Matrix(const Matrix &other, int start_rowid, int end_rowid)
    : rows(end_rowid - start_rowid), cols(other.cols),
      data(other.at(start_rowid * other.cols)), storage(other.storage),
      itemsize(other.itemsize) {
  if (end_rowid < start_rowid) {
    throw std::invalid_argument("end_rowid < start_rowid for matrix slice");
  }
  if (end_rowid > other.rows) {
    throw std::invalid_argument("row index out of bounds for matrix");
  }
}

template <typename T>
void copy_rowids(const T *input, const int *rowids, int rows, int cols,
                 T *output) {
  // copy rows over
  auto count = thrust::make_counting_iterator<int>(0);
  thrust::for_each(count, count + (rows * cols), [=] __device__(int i) {
    int col = i % cols;
    int row = rowids[i / cols];
    output[i] = input[col + row * cols];
  });
}

Matrix::Matrix(const Matrix &other, const Vector<int> &rowids)
    : rows(rowids.size), cols(other.cols), itemsize(other.itemsize) {
  storage.reset(
      new rmm::device_buffer(itemsize * rows * cols, rmm::cuda_stream_view()));
  data = storage->data();
  // TODO:
  if (itemsize == 4) {
    copy_rowids<float>(other, rowids.data, rows, cols, *this);
  } else {
    throw std::runtime_error("unknown itemsize initializing Matrix");
  }
}

Matrix::Matrix(int rows, int cols, void *host_data, bool allocate, int itemsize)
    : rows(rows), cols(cols), itemsize(itemsize) {
  if (allocate) {
    storage.reset(new rmm::device_buffer(itemsize * rows * cols,
                                         rmm::cuda_stream_view()));
    data = storage->data();
    if (host_data) {
      CHECK_CUDA(hipMemcpy(data, host_data, rows * cols * itemsize,
                            hipMemcpyHostToDevice));
    }
  } else {
    data = host_data;
  }
}

void Matrix::resize(int rows, int cols) {
  if (cols != this->cols) {
    throw std::logic_error(
        "changing number of columns in Matrix::resize is not implemented yet");
  }
  if (rows < this->rows) {
    throw std::logic_error(
        "reducing number of rows in Matrix::resize is not implemented yet");
  }
  auto new_storage =
      new rmm::device_buffer(itemsize * rows * cols, rmm::cuda_stream_view());
  CHECK_CUDA(hipMemcpy(new_storage->data(), data,
                        this->rows * this->cols * itemsize,
                        hipMemcpyDeviceToDevice));
  int extra_rows = rows - this->rows;
  storage.reset(new_storage);
  data = storage->data();
  CHECK_CUDA(
      hipMemset(at(this->rows * this->cols), 0, extra_rows * cols * itemsize));

  this->rows = rows;
  this->cols = cols;
}

void Matrix::assign_rows(const Vector<int> &rowids, const Matrix &other) {
  if (other.cols != cols) {
    throw std::invalid_argument(
        "column dimensionality mismatch in Matrix::assign_rows");
  }

  auto count = thrust::make_counting_iterator<int>(0);
  int other_cols = other.cols, other_rows = other.rows;

  int *rowids_data = rowids.data;

  const float *other_data = other;
  float *self_data = *this;

  thrust::for_each(count, count + (other_rows * other_cols),
                   [=] __device__(int i) {
                     int col = i % other_cols;
                     int row = rowids_data[i / other_cols];
                     int idx = col + row * other_cols;
                     self_data[idx] = other_data[i];
                   });
}

template <typename T>
__global__ void calculate_norms_kernel(const T *input, int rows, int cols,
                                       T *output) {
  static __shared__ float shared[32];
  for (int i = blockIdx.x; i < rows; i += gridDim.x) {
    float value = convert<T, float>(input[i * cols + threadIdx.x]);
    float squared_norm = dot(value, value, shared);
    if (threadIdx.x == 0) {
      float norm = sqrt(squared_norm);
      if (norm == 0) {
        norm = 1e-10;
      }
      output[i] = convert<float, T>(norm);
    }
  }
}

Matrix Matrix::calculate_norms() const {
  int devId;
  CHECK_CUDA(hipGetDevice(&devId));

  int multiprocessor_count;
  CHECK_CUDA(hipDeviceGetAttribute(&multiprocessor_count,
                                    hipDeviceAttributeMultiprocessorCount, devId));

  int block_count = 256 * multiprocessor_count;
  int thread_count = cols;

  Matrix output(1, rows, NULL);

  if (itemsize == 4) {
    calculate_norms_kernel<float>
        <<<block_count, thread_count>>>(*this, rows, cols, output);
    // TODO  } else if (itemsize == 2) {
    //    calculate_norms_kernel<half><<<block_count, thread_count>>>(
    //        data, rows, cols, output.data);
  } else {
    throw std::runtime_error("unknown itemsize in calculate_norms");
  }

  CHECK_CUDA(hipDeviceSynchronize());
  return output;
}

void Matrix::to_host(void *out) const {
  CHECK_CUDA(
      hipMemcpy(out, data, rows * cols * itemsize, hipMemcpyDeviceToHost));
}

CSRMatrix::CSRMatrix(int rows, int cols, int nonzeros, const int *indptr_,
                     const int *indices_, const float *data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

  CHECK_CUDA(hipMalloc(&indptr, (rows + 1) * sizeof(int)));
  CHECK_CUDA(hipMemcpy(indptr, indptr_, (rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&indices, nonzeros * sizeof(int)));
  CHECK_CUDA(hipMemcpy(indices, indices_, nonzeros * sizeof(int),
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
  CHECK_CUDA(
      hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

CSRMatrix::~CSRMatrix() {
  CHECK_CUDA(hipFree(indices));
  CHECK_CUDA(hipFree(indptr));
  CHECK_CUDA(hipFree(data));
}

COOMatrix::COOMatrix(int rows, int cols, int nonzeros, const int *row_,
                     const int *col_, const float *data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

  CHECK_CUDA(hipMalloc(&row, nonzeros * sizeof(int)));
  CHECK_CUDA(
      hipMemcpy(row, row_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&col, nonzeros * sizeof(int)));
  CHECK_CUDA(
      hipMemcpy(col, col_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
  CHECK_CUDA(
      hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

COOMatrix::~COOMatrix() {
  CHECK_CUDA(hipFree(row));
  CHECK_CUDA(hipFree(col));
  CHECK_CUDA(hipFree(data));
}
} // namespace gpu
} // namespace implicit

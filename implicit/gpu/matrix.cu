#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>

#include "implicit/gpu/convert.cuh"
#include "implicit/gpu/dot.cuh"
#include "implicit/gpu/matrix.h"
#include "implicit/gpu/utils.h"

namespace implicit {
namespace gpu {
template <typename T>
Vector<T>::Vector(int size, const T *host_data)
    : size(size),
      storage(new rmm::device_uvector<T>(size, rmm::cuda_stream_view())),
      data(storage->data()) {
  if (host_data) {
    CHECK_CUDA(
        hipMemcpy(data, host_data, size * sizeof(T), hipMemcpyHostToDevice));
  }
}

template <typename T> void Vector<T>::to_host(T *out) const {
  CHECK_CUDA(hipMemcpy(out, data, size * sizeof(T), hipMemcpyDeviceToHost));
}

template struct Vector<char>;
template struct Vector<int>;
template struct Vector<float>;

template <typename T>
Matrix<T>::Matrix(const Matrix<T> &other, int rowid)
    : rows(1), cols(other.cols), data(other.data + rowid * other.cols),
      storage(other.storage) {
  if (rowid >= other.rows) {
    throw std::invalid_argument("row index out of bounds for matrix");
  }
}

template <typename T>
Matrix<T>::Matrix(const Matrix<T> &other, int start_rowid, int end_rowid)
    : rows(end_rowid - start_rowid), cols(other.cols),
      data(other.data + start_rowid * other.cols), storage(other.storage) {
  if (end_rowid < start_rowid) {
    throw std::invalid_argument("end_rowid < start_rowid for matrix slice");
  }
  if (end_rowid > other.rows) {
    throw std::invalid_argument("row index out of bounds for matrix");
  }
}

template <typename T>
void copy_rowids(const T *input, const int *rowids, int rows, int cols,
                 T *output) {
  // copy rows over
  auto count = thrust::make_counting_iterator<int>(0);
  thrust::for_each(count, count + (rows * cols), [=] __device__(int i) {
    int col = i % cols;
    int row = rowids[i / cols];
    output[i] = input[col + row * cols];
  });
}

template <typename T>
Matrix<T>::Matrix(const Matrix<T> &other, const Vector<int> &rowids)
    : rows(rowids.size), cols(other.cols) {
  storage.reset(
      new rmm::device_uvector<T>(rows * cols, rmm::cuda_stream_view()));
  data = storage->data();
  copy_rowids(other.data, rowids.data, rows, cols, data);
}

template <typename T>
Matrix<T>::Matrix(int rows, int cols, T *host_data, bool allocate)
    : rows(rows), cols(cols) {
  if (allocate) {
    storage.reset(
        new rmm::device_uvector<T>(rows * cols, rmm::cuda_stream_view()));
    data = storage->data();
    if (host_data) {
      CHECK_CUDA(hipMemcpy(data, host_data, rows * cols * sizeof(T),
                            hipMemcpyHostToDevice));
    }
  } else {
    data = host_data;
  }
}

template <typename T>
void Matrix<T>::resize(int rows, int cols) {
  if (cols != this->cols) {
    throw std::logic_error(
        "changing number of columns in Matrix::resize is not implemented yet");
  }
  if (rows < this->rows) {
    throw std::logic_error(
        "reducing number of rows in Matrix::resize is not implemented yet");
  }
  auto new_storage =
      new rmm::device_uvector<T>(rows * cols, rmm::cuda_stream_view());
  CHECK_CUDA(hipMemcpy(new_storage->data(), data,
                        this->rows * this->cols * sizeof(T),
                        hipMemcpyDeviceToDevice));
  int extra_rows = rows - this->rows;
  CHECK_CUDA(hipMemset(new_storage->data() + this->rows * this->cols, 0,
                        extra_rows * cols * sizeof(T)));
  storage.reset(new_storage);
  data = storage->data();
  this->rows = rows;
  this->cols = cols;
}

template <typename T>
void Matrix<T>::assign_rows(const Vector<int> &rowids, const Matrix<T> &other) {
  if (other.cols != cols) {
    throw std::invalid_argument(
        "column dimensionality mismatch in Matrix::assign_rows");
  }

  auto count = thrust::make_counting_iterator<int>(0);
  int other_cols = other.cols, other_rows = other.rows;

  int *rowids_data = rowids.data;
  T *other_data = other.data;
  T *self_data = data;

  thrust::for_each(count, count + (other_rows * other_cols),
                   [=] __device__(int i) {
                     int col = i % other_cols;
                     int row = rowids_data[i / other_cols];
                     int idx = col + row * other_cols;
                     self_data[idx] = other_data[i];
                   });
}

template <typename T>
__global__ void calculate_norms_kernel(const T *input, int rows, int cols,
                                       T *output) {
  static __shared__ float shared[32];
  for (int i = blockIdx.x; i < rows; i += gridDim.x) {
    float value = convert<T, float>(input[i * cols + threadIdx.x]);
    float squared_norm = dot(value, value, shared);
    if (threadIdx.x == 0) {
      float norm = sqrt(squared_norm);
      if (norm == 0) {
        norm = 1e-10;
      }
      output[i] = convert<float, T>(norm);
    }
  }
}

template <typename T>
Matrix<T> Matrix<T>::calculate_norms() const {
  int devId;
  CHECK_CUDA(hipGetDevice(&devId));

  int multiprocessor_count;
  CHECK_CUDA(hipDeviceGetAttribute(&multiprocessor_count,
                                    hipDeviceAttributeMultiprocessorCount, devId));

  int block_count = 256 * multiprocessor_count;
  int thread_count = cols;

  Matrix<T> output(1, rows, NULL);
  calculate_norms_kernel<<<block_count, thread_count>>>(
      data, rows, cols, output.data);

  CHECK_CUDA(hipDeviceSynchronize());
  return output;
}

template <typename T>
void Matrix<T>::to_host(T *out) const {
  CHECK_CUDA(hipMemcpy(out, data, rows * cols * sizeof(T),
                        hipMemcpyDeviceToHost));
}

template struct Matrix<float>;
template struct Matrix<half>;

CSRMatrix::CSRMatrix(int rows, int cols, int nonzeros, const int *indptr_,
                     const int *indices_, const float *data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

  CHECK_CUDA(hipMalloc(&indptr, (rows + 1) * sizeof(int)));
  CHECK_CUDA(hipMemcpy(indptr, indptr_, (rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&indices, nonzeros * sizeof(int)));
  CHECK_CUDA(hipMemcpy(indices, indices_, nonzeros * sizeof(int),
                        hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
  CHECK_CUDA(
      hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

CSRMatrix::~CSRMatrix() {
  CHECK_CUDA(hipFree(indices));
  CHECK_CUDA(hipFree(indptr));
  CHECK_CUDA(hipFree(data));
}

COOMatrix::COOMatrix(int rows, int cols, int nonzeros, const int *row_,
                     const int *col_, const float *data_)
    : rows(rows), cols(cols), nonzeros(nonzeros) {

  CHECK_CUDA(hipMalloc(&row, nonzeros * sizeof(int)));
  CHECK_CUDA(
      hipMemcpy(row, row_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&col, nonzeros * sizeof(int)));
  CHECK_CUDA(
      hipMemcpy(col, col_, nonzeros * sizeof(int), hipMemcpyHostToDevice));

  CHECK_CUDA(hipMalloc(&data, nonzeros * sizeof(float)));
  CHECK_CUDA(
      hipMemcpy(data, data_, nonzeros * sizeof(int), hipMemcpyHostToDevice));
}

COOMatrix::~COOMatrix() {
  CHECK_CUDA(hipFree(row));
  CHECK_CUDA(hipFree(col));
  CHECK_CUDA(hipFree(data));
}
} // namespace gpu
} // namespace implicit
